#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, const float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

int main() {
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N * sizeof(float));
  y = (float*)malloc(N * sizeof(float));
  for (int i = 0; i < N; i++) { x[i] = 1.0f; y[i] = 2.0f; }
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  printf("y[0]=%f y[N-1]=%f\n", y[0], y[N-1]);
  hipFree(d_x); hipFree(d_y);
  free(x); free(y);
  return 0;
}
